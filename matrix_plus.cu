#include <stdio.h>
#include<hip/hip_runtime.h>


const int N = 100 ;//256

__global__ void MatrixAdd_CUDA(int *A, int *B, int *C) { 
      int i= blockIdx.y*blockDim.y+ threadIdx.y; 
      int j = blockIdx.x*blockDim.x+ threadIdx.x; 
    *(C + i*N + j) =  *(A + i*N + j)+ *(B + i*N + j); 
      
}      
void DisplayMatrix(int *A, int row,  int col)
{
    int i,j;
    for(i=0;i<row;i++){
        for(j=0;j<col;j++) printf("  %d",*(A+i*col+j));
        printf("\n");
    }
}
   
int main (void)
{
      int *Host_a, *Host_b, *Host_c;
      Host_a  = (int *) malloc ((N*N)*sizeof(int));
      Host_b   = (int *) malloc ((N*N)*sizeof(int));
      Host_c   = (int *) malloc ((N*N)*sizeof(int));
     

      int *dev_a , *dev_b, *dev_c ;
       hipMalloc(&dev_a , (N*N)*sizeof(int));
      hipMalloc(&dev_b , (N*N)*sizeof(int));
      hipMalloc(&dev_c , (N*N)*sizeof(int));

      for ( int i = 0; i <N ; i++ )
          for(int j=0;j<N;j++){
              *(Host_a+i*N+j)=i*2+1;
              *(Host_b+i*N+j)=i+j;
               
          }
      hipMemcpy (dev_a , Host_a , (N*N)*sizeof(int) , hipMemcpyHostToDevice);
      hipMemcpy (dev_b , Host_b , (N*N)*sizeof(int) , hipMemcpyHostToDevice);

      //int threadsPerBlock = 256;    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock
   
      dim3 threadsPerBlock(10, 10);    //16 16 
      dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y); 

      MatrixAdd_CUDA <<< numBlocks, threadsPerBlock  >>> (dev_a , dev_b , dev_c ) ;

     hipMemcpy(Host_c , dev_c , (N*N)*sizeof(int) , hipMemcpyDeviceToHost);

      DisplayMatrix(Host_c,10,10);
                 

      hipFree (dev_a) ;
      hipFree (dev_b) ;
      hipFree (dev_c) ;
      
      return 0 ;

}