
#include <hip/hip_runtime.h>
#include <stdio.h>

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ))

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
      {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
        exit( EXIT_FAILURE );
    }
}


const int N = 10000 ;

// CUDA Kernel
__global__ void Vector_Plus ( int *AG ,  int *BG , int *CG)
{
      int id = blockDim.x*blockIdx.x+threadIdx.x ;
      if ( id < N )
            *(CG+id)=*(AG+id)+ *(BG+id);

}


int main (void)
{

      //Khoi tao 3 mang A B C tren CPU
      int *A, *B, *C;
      A   = (int *) malloc (N*sizeof(int));
      B   = (int *) malloc (N*sizeof(int));
      C   = (int *) malloc (N*sizeof(int));

   

      //Khoi tao 3 mang A B C tren GPU
      int *AG , *BG, *CG ;
      HANDLE_ERROR ( hipMalloc(&AG , N*sizeof(int) ) );
      HANDLE_ERROR ( hipMalloc(&BG , N*sizeof(int) ) );
      HANDLE_ERROR ( hipMalloc(&CG , N*sizeof(int) ) );

      //Khoi tao gia tri mang A B tren CPU
      for ( int i = 0; i <N ; i++ )
      {
            *(A+i) = i ;
            *(B+i) = i+1 ; 
      }

      //Copy mang A B  sang GPU
      HANDLE_ERROR (hipMemcpy (AG , A , N*sizeof(int) , hipMemcpyHostToDevice));
      HANDLE_ERROR (hipMemcpy (BG , B , N*sizeof(int) , hipMemcpyHostToDevice));

      
      int threadsPerBlock = 1000;    
      int blocksPerGrid = N / threadsPerBlock;
      //Vector_Plus <<< 1, N  >>> (AG , BG , CG ) ;
      Vector_Plus <<<blocksPerGrid, threadsPerBlock >>> (AG , BG , CG ) ;

      //Copy lai CPU
      HANDLE_ERROR (hipMemcpy(C , CG , N*sizeof(int) , hipMemcpyDeviceToHost));

      //Hien thi ket qua
      for ( int i = 0; i<N; i++ )
            printf ("%d + %d = %d\n", *(A+i) , *(B+i) , *(C+i)) ;

      //Gia phong bo nho
      hipFree (AG) ;
      hipFree (BG) ;
      hipFree (CG) ;

      system("pause");
      return 0 ;

}