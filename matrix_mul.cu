
#include <hip/hip_runtime.h>
#include <stdio.h>

const int m=20;
const int n=20;
const int p=5;

//m*n *n*p=m*p

__global__ void MatrixAdd_CUDA(int *A, int *B, int *C) { 
      int row = blockIdx.y * blockDim.y + threadIdx.y;    
      int col = blockIdx.x * blockDim.x + threadIdx.x;
      int s ;
      if(row<m && col<p){
            s=  0; 
            for (int k = 0 ; k < n ; k++ ) {
                  s +=*(A + row*n + k)*(*(B + k*p + col));
            }
            *(C+row*p+col) = s;
      }
     
          

      
}      
    
void DisplayMatrix(int *A, int row,  int col)
{
    int i,j;
    for(i=0;i<row;i++){
        for(j=0;j<col;j++) printf("  %d",*(A+i*col+j));
        printf("\n");
    }
}

int main (void)
{
      int *A, *B, *C;
      A  = (int *) malloc ((m*n)*sizeof(int));
      B   = (int *) malloc ((n*p)*sizeof(int));
      C   = (int *) malloc ((m*p)*sizeof(int));
     

      int *AG , *BG, *CG ;
      
      hipMalloc(&AG , (m*n)*sizeof(int) ) ;
      hipMalloc(&BG , (m*n)*sizeof(int) ) ;
      hipMalloc(&CG , (m*p)*sizeof(int) ) ;

      for ( int i = 0; i <m ; i++ )
          for(int j=0;j<n;j++){
                if(i==j){
                  *(A+i*n+j)=1;
                }else{
                  *(A+i*n+j)=0;
                } 
          }
      for ( int i = 0; i <n ; i++ )
          for(int j=0;j<p;j++){
              *(B+i*p+j)=i+j;    
          }

      hipMemcpy (AG , A , (m*n)*sizeof(int) , hipMemcpyHostToDevice);
      hipMemcpy (BG , B , (n*p)*sizeof(int) , hipMemcpyHostToDevice);

      dim3 threadsPerBlock(5, 5);    
      dim3 numBlocks(m / threadsPerBlock.x, m / threadsPerBlock.y); 
      MatrixAdd_CUDA <<< numBlocks, threadsPerBlock  >>> (AG , BG , CG ) ;

      hipMemcpy(C , CG , (m*p)*sizeof(int) , hipMemcpyDeviceToHost);

      DisplayMatrix(A,m,n);
      printf("======================================================================================\n");
      DisplayMatrix(B,n,p);

      printf("=======================================================================================\n");
      DisplayMatrix(C,m,p);

                 

      hipFree (AG) ;
      hipFree (BG) ;
      hipFree (CG) ;
      free(A);
      free(B);
      free(C);

    
      return 0 ;

}