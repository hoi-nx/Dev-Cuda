
#include <hip/hip_runtime.h>
#include <stdio.h>


const int N = 10000 ;

__global__ void Vector_Addition (  int *dev_a ,  int *dev_b , int *dev_c)
{ 
      //Lay ra id cua thread trong 1 block.
      int tid = blockIdx.x ; // blockDim.x*blockIdx.x+threadIdx.x
     
      if ( tid < N )
            *(dev_c+tid) = *(dev_a+tid) + *(dev_b+tid) ;

}


int main (void)
{

      //Cap phat bo nho 3 mang A B C tren CPU
      int *Host_a, *Host_b, *Host_c;
      Host_a  = (int *) malloc (N*sizeof(int));
      Host_b   = (int *) malloc (N*sizeof(int));
      Host_c   = (int *) malloc (N*sizeof(int));
      //Khởi tao bên CPU
      for ( int i = 0; i <N ; i++ )
      {
            *(Host_a+i) = i ;
            *(Host_b+i) = i+1 ; 
      }

	  //Cap phat bo nho 3 mang A B C tren GPU
      int *dev_a , *dev_b, *dev_c ;
      hipMalloc(&dev_a , N*sizeof(int) ) ;
      hipMalloc(&dev_b , N*sizeof(int) ) ;
      hipMalloc(&dev_c , N*sizeof(int) ) ;

     
      //Copy mang host_a, host_b tu CPU cho mang dev_a,dev_b tren GPU
      hipMemcpy (dev_a , Host_a , N*sizeof(int) , hipMemcpyHostToDevice);
      hipMemcpy (dev_b , Host_b , N*sizeof(int) , hipMemcpyHostToDevice);

      //Tính toán trên GPU
	  //N block/gird ,1 thread/1 block
      Vector_Addition <<< N, 1  >>> (dev_a , dev_b , dev_c ) ;

      //Copy lại CPU
      hipMemcpy(Host_c , dev_c , N*sizeof(int) , hipMemcpyDeviceToHost);

      //Ket qua
      for ( int i = 0; i<N; i++ )
                  printf ("%d + %d = %d\n", *(Host_a+i) , *(Host_b+i)  , *(Host_c+i)  ) ;

      //Gia phong bo nhe
      hipFree (dev_a) ;
      hipFree (dev_b) ;
      hipFree (dev_c) ;

      system("pause");
      return 0 ;

}